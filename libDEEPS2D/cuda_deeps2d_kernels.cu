
#include <hip/hip_runtime.h>
/*******************************************************************************
*   OpenHyperFLOW2D-CUDA                                                       *
*                                                                              *
*   Transient, Density based Effective Explicit Parallel Hybrid Solver         *
*   TDEEPHS (CUDA+MPI)                                                         *
*   Version  1.0.1                                                             *
*   Copyright (C)  1995-2014 by Serge A. Suchkov                               *
*   Copyright policy: LGPL V3                                                  *
*                                                                              *
*   deeps2d_core.cpp: CUDA kernels code.                                       *
*                                                                              *
*  last update: 16/01/2014                                                     *
********************************************************************************/

#ifdef _CUDA_
#define _PARALLEL_ONLY

#include "libDEEPS2D/deeps2d_core.hpp"

#ifdef _CUDA_
__host__
#endif //_CUDA_ 
int LoadTable2GPU(Table* Src, Table*& Dst, int i_dev)
{
 Table* pTmpTable;
 pTmpTable = new Table(NULL,0); 
 pTmpTable->n = Src->n;
 
 if(cudaSetDevice(i_dev) != cudaSuccess ) {
    printf("\nError set CUDA device no: %d\n",i_dev);
    Exit_OpenHyperFLOW2D(1);
 }

 if(cudaMalloc( (void**)&pTmpTable->x, sizeof(double)*Src->n ) == cudaErrorMemoryAllocation) {
    printf("\nError allocate GPU memory for Table %s\n",Src->GetName());
    Exit_OpenHyperFLOW2D(1);
 }

 if(cudaMalloc( (void**)&pTmpTable->y, sizeof(double)*Src->n ) == cudaErrorMemoryAllocation) {
   printf("\nError allocate GPU memory for Table %s",Src->GetName());
   Exit_OpenHyperFLOW2D(1);
 }

 CopyHostToDevice(Src->x,pTmpTable->x,sizeof(double)*Src->n); 
 CopyHostToDevice(Src->y,pTmpTable->y,sizeof(double)*Src->n);
 
 if(cudaMalloc( (void**)&Dst, sizeof(Table) ) == cudaErrorMemoryAllocation) {
   printf("\nError allocate GPU memory for Table %s\n",Src->GetName());
   Exit_OpenHyperFLOW2D(1);
 }

 CopyHostToDevice(pTmpTable,Dst,sizeof(Table));
 
 //delete pTmpTable;
 
 return Src->n;
 
}


#ifdef _CUDA_
 __host__ __device__
#endif //_CUDA_ 
inline double GetVal(Table* t,
                     double _x ) {
    if ( !t )
        return 0.;
    
    register int  i, _n = t->n;
    
    register double _y;

    if ( _n == 1 )
        return( t->y[0] );

    if ( _x <= t->x[0] ) {
        i = 1;
        goto EndGetVal;
    }

    if ( _x >= t->x[t->n-1] ) {
        i = _n - 1;
        goto EndGetVal;
    }

    for ( i=1; i<_n; i++ ) {
        if ( (_x >= t->x[i-1]) && (_x < t->x[i]) )
            break;
    }

    EndGetVal:

    _y = t->y[i] + (t->y[i-1] - t->y[i])*(_x - t->x[i])/(t->x[i-1] - t->x[i]);

    return( _y );
}

#ifdef _CUDA_
 __host__ __device__
#endif //_CUDA_ 
int cuda_CalcChemicalReactions(FlowNode2D<double,NUM_COMPONENTS>* CalcNode,
                               ChemicalReactionsModel cr_model, void* CRM_data) {
    
    ChemicalReactionsModelData2D* model_data = (ChemicalReactionsModelData2D*)CRM_data;
    double   Y0,Yfu,Yox,Ycp,Yair;

    Yfu  = CalcNode->S[i2d_Yfu]/CalcNode->S[0]; // Fuel
    Yox  = CalcNode->S[i2d_Yox]/CalcNode->S[0]; // OX
    Ycp  = CalcNode->S[i2d_Ycp]/CalcNode->S[0]; // cp
    Yair = 1. - (Yfu+Yox+Ycp);                  // air

    if(cr_model==CRM_ZELDOVICH) {
//--- chemical reactions (Zeldovich model) -------------------------------------------------->
      if ( !CalcNode->isCond2D(CT_Y_CONST_2D) ) {
          Y0   = 1./(Yfu+Yox+Ycp+Yair);
          Yfu  = Yfu*Y0;
          Yox  = Yox*Y0;
          Ycp  = Ycp*Y0;

          if ( CalcNode->Tg > CalcNode->Tf ) {
              if ( Yox > Yfu*model_data->K0 ) { // Yo2 > Yfuel
                   Yox = Yox - Yfu*model_data->K0;
                   Yfu = 0.;
                   Ycp = 1.-Yox-Yair;
              } else {                          // Yo2 < Yfuel
                   Yfu = Yfu - Yox/model_data->K0;
                   Yox = 0.;
                   Ycp = 1. -Yfu-Yair;
              }
           }
        }
//--- chemical reactions (Zeldovich model) -------------------------------------------------->
    }

    CalcNode->R   = model_data->R_Fuel*Yfu+
                            model_data->R_OX*Yox+
                            model_data->R_cp*Ycp+
                            model_data->R_air*Yair;
    CalcNode->mu  = GetVal(model_data->mu_Fuel,CalcNode->Tg)*Yfu+
                    GetVal(model_data->mu_OX,CalcNode->Tg)*Yox+
                    GetVal(model_data->mu_cp,CalcNode->Tg)*Ycp+
                    GetVal(model_data->mu_air,CalcNode->Tg)*Yair;
    CalcNode->CP  = GetVal(model_data->Cp_Fuel,CalcNode->Tg)*Yfu+
                    GetVal(model_data->Cp_OX,CalcNode->Tg)*Yox+
                    GetVal(model_data->Cp_cp,CalcNode->Tg)*Ycp+
                    GetVal(model_data->Cp_air,CalcNode->Tg)*Yair;
    CalcNode->lam = GetVal(model_data->lam_Fuel,CalcNode->Tg)*Yfu+
                    GetVal(model_data->lam_OX,CalcNode->Tg)*Yox+
                    GetVal(model_data->lam_cp,CalcNode->Tg)*Ycp+
                    GetVal(model_data->lam_air,CalcNode->Tg)*Yair;

    if ( Yair<1.e-8 ) {
         Yair =0.;
      }
    if ( Ycp<1.e-8 ) {
         Ycp =0.;
      }
    if ( Yox<1.e-8 ) {
         Yox =0.;
      }
    if ( Yfu<1.e-8 ) {
         Yfu =0.;
      }

     Y0   = 1./(Yfu+Yox+Ycp+Yair);
     Yfu  = Yfu*Y0;
     Yox  = Yox*Y0;
     Ycp  = Ycp*Y0;
     Yair = Yair*Y0;


    CalcNode->Y[0] = Yfu;
    CalcNode->Y[1] = Yox;
    CalcNode->Y[2] = Ycp;
    CalcNode->Y[3] = Yair;

    if ( !CalcNode->isCond2D(CT_Y_CONST_2D) ) {
          CalcNode->S[i2d_Yfu] = fabs(Yfu*CalcNode->S[0]);
          CalcNode->S[i2d_Yox] = fabs(Yox*CalcNode->S[0]);
          CalcNode->S[i2d_Ycp] = fabs(Ycp*CalcNode->S[0]);
     }
 return 1;
}

void SetP2PAccess(int dev1, int dev2) {
 cudaError_t cudaState;
 int canAccess = 0;

 cudaState = cudaDeviceCanAccessPeer(&canAccess,dev1,dev2);

 if(cudaState != cudaSuccess) {
     printf("\nError set P2P access for devices %i<-->%i\n",dev1,dev2);
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
     Exit_OpenHyperFLOW2D(1);
 }

 if(!canAccess) {
     cudaSetDevice(dev1);
     cudaState = cudaDeviceEnablePeerAccess(dev2,0);

     if(cudaState != cudaSuccess) {
         printf("\nError set P2P access for devices %i<-->%i\n",dev1,dev2);
         printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
         Exit_OpenHyperFLOW2D(1);
     }
 }

 canAccess = 0;

 cudaSetDevice(dev2);

 cudaState = cudaDeviceCanAccessPeer(&canAccess,dev2,dev1);

 if(cudaState != cudaSuccess) {
    printf("\nError set P2P access for devices %i<-->%i\n",dev2,dev1);
    printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
    Exit_OpenHyperFLOW2D(1);
 }
 
 if(!canAccess) {
     cudaSetDevice(dev2);
     cudaState = cudaDeviceEnablePeerAccess(dev1,0);

     if(cudaState != cudaSuccess) {
         printf("\nError set P2P access for devices %i<-->%i\n",dev2,dev1);
         printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
         Exit_OpenHyperFLOW2D(1);
     }
 }
}


void DisableP2PAccess(int dev1, int dev2) {
 cudaError_t cudaState;
 int canAccess = 0;

 cudaState = cudaDeviceCanAccessPeer(&canAccess,dev1,dev2);

 if(cudaState != cudaSuccess) {
     printf("\nError set P2P access for devices %i<-->%i\n",dev1,dev2);
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
     Exit_OpenHyperFLOW2D(1);
 }

 if(canAccess) {
     cudaSetDevice(dev1);
     cudaState = cudaDeviceDisablePeerAccess(dev2);

     if(cudaState != cudaSuccess) {
         printf("\nError set P2P access for devices %i<-->%i\n",dev1,dev2);
         printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
         Exit_OpenHyperFLOW2D(1);
     }
 }

 canAccess = 0;

 cudaSetDevice(dev2);

 cudaState = cudaDeviceCanAccessPeer(&canAccess,dev2,dev1);

 if(cudaState != cudaSuccess) {
    printf("\nError set P2P access for devices %i<-->%i\n",dev2,dev1);
    printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
    Exit_OpenHyperFLOW2D(1);
 }

 if(canAccess) {
     cudaSetDevice(dev2);
     cudaState = cudaDeviceDisablePeerAccess(dev1);

     if(cudaState != cudaSuccess) {
         printf("\nError set P2P access for devices %i<-->%i\n",dev2,dev1);
         printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
         Exit_OpenHyperFLOW2D(1);
     }
 }
}

void CUDA_BARRIER(char* KernelName) {
    cudaError_t cudaState = cudaDeviceSynchronize();    
    if(cudaState != cudaSuccess) {
        printf("\nError in %s kernel...\n",KernelName);
        printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
        Exit_OpenHyperFLOW2D(1);
    }
}

void CopyDeviceToDeviceP2P(void* src, int src_dev,
                           void* dst, int dst_dev,
                           size_t length, cudaStream_t 	cuda_stream) {
    cudaError_t cudaState = cudaMemcpyPeerAsync(dst, dst_dev, src, src_dev, length, cuda_stream); 
    if(cudaState != cudaSuccess) {
     printf("\nError P2P copy device to device...\n");
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );

       Exit_OpenHyperFLOW2D(1);
    }
}

void CopyDeviceToDevice(void* src, void* dst, size_t length, cudaStream_t stream) {
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyDeviceToDevice, stream);
    if(cudaState != cudaSuccess) {
     printf("\nError copy device to device...\n");
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );

       Exit_OpenHyperFLOW2D(1);
    }
}

void CopyHostToDevice(void* src, void* dst, size_t length, cudaStream_t stream) {
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyHostToDevice, stream);
    if(cudaState != cudaSuccess) {
     printf("\nError copy host to device...\n");
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
       Exit_OpenHyperFLOW2D(1);
    }
}

void CopyDeviceToHost(void* src, void* dst, size_t length, cudaStream_t stream) {
    cudaError_t cudaState = cudaMemcpyAsync(dst, src, length,cudaMemcpyDeviceToHost, stream);
    if(cudaState != cudaSuccess) {
     printf("\nError copy device to host...\n");
     printf("%s\n", cudaGetErrorString( cudaGetLastError() ) );
       Exit_OpenHyperFLOW2D(1);
    }
}

__global__ void 
cuda_SetInitBoundaryLayer(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                          unsigned long int index_limit,
                          int X0, int MAX_Y,
                          double delta,
                          double sig_w, 
                          double sig_f,
                          TurbulenceExtendedModel etm,
                          double _dx, double _dy,
                          double* _Hu,
                          int _isSrcAdd,
                          FlowType _FT,
                          SolverMode sm) {

    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

              FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pJ2D[index];

              if(CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {

                  if(CurrentNode->time == 0. &&  delta > 0.0  &&  CurrentNode->l_min <= delta) {
                     CurrentNode->S[i2d_RoU] = CurrentNode->S[i2d_RoU] * CurrentNode->l_min/delta;
                     CurrentNode->S[i2d_RoV] = CurrentNode->S[i2d_RoV] * CurrentNode->l_min/delta;
                     CurrentNode->FillNode2D(0,1,sig_w,sig_f,etm,delta,_dx,_dy,_Hu,_isSrcAdd,sm,_FT);
                  }

               if(CurrentNode->CT != (ulong)(NT_FC_2D)) {

                  int  i = CurrentNode->ix - X0;
                  int  j = CurrentNode->iy;

                  int  n1 = CurrentNode->idXl;
                  int  n2 = CurrentNode->idXr;
                  int  n3 = CurrentNode->idYu;
                  int  n4 = CurrentNode->idYd;

                  int  N1 = i - n1;
                  int  N2 = i + n2;
                  int  N3 = j + n3;
                  int  N4 = j - n4;

                  CurrentNode->UpNode    = &pJ2D[i*MAX_Y + N3];
                  CurrentNode->DownNode  = &pJ2D[i*MAX_Y + N4];
                  CurrentNode->RightNode = &pJ2D[N2*MAX_Y + j];
                  CurrentNode->LeftNode  = &pJ2D[N1*MAX_Y + j];
               }
          }
   }
}

__global__ void
cuda_SetMinDistanceToWall2D(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                            unsigned long int index_limit,
                            XY<int>* WallNodes2D, 
                            int NumWallNodes2D,
                            double min_l_min,
                            double max_l_min,
                            double _dx, double _dy)   {

   unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

   if(index < index_limit) {

       FlowNode2D<double,NUM_COMPONENTS>* TmpNode = &pJ2D[index];

       if(TmpNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {   

           TmpNode->l_min = max_l_min;
 #pragma unroll
           for (int ii=0;ii<NumWallNodes2D;ii++) {

              XY<int>*  TmpWallNode = &WallNodes2D[ii]; 

              double L_x   = (TmpWallNode->X - TmpNode->ix)* _dx;
              double L_y   = (TmpWallNode->Y - TmpNode->iy)* _dy;
              double l_min = sqrt(L_x*L_x + L_y*L_y);

              TmpNode->l_min = max(min(TmpNode->l_min,l_min),min_l_min);
            }
        }
   }
   //__syncthreads();
}
__global__ void 
cuda_Recalc_y_plus(FlowNode2D<double,NUM_COMPONENTS>* pJ2D,
                   unsigned long int index_limit,
                   XY<int>* WallNodes2D,
                   int NumWallNodes2D,
                   double min_l_min,
                   double max_l_min,
                   double _dx, 
                   double _dy,
                   int max_y) {

    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

        FlowNode2D<double,NUM_COMPONENTS>* TmpNode = &pJ2D[index];

        if(TmpNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D)) {

            int iw=0;
            int jw=0;
#pragma unroll
           for (int ii=0;ii<NumWallNodes2D;ii++) {

               XY<int>*  TmpWallNode = &WallNodes2D[ii]; 

               double L_x   = (TmpWallNode->X - TmpNode->ix)* _dx;
               double L_y   = (TmpWallNode->Y - TmpNode->iy)* _dy;
               double l_min = sqrt(L_x*L_x + L_y*L_y);

               if(l_min  == TmpNode->l_min) {
                  iw = TmpWallNode->X;
                  jw = TmpWallNode->Y;
               }
           }

          //__syncthreads();

           unsigned long int wall_index = iw*max_y + jw;

           if(wall_index < index_limit) {

               FlowNode2D<double,NUM_COMPONENTS>* WallNode = &pJ2D[wall_index];   // x*nY + y

               double tau_w = (fabs(WallNode->dUdy) +
                               fabs(WallNode->dVdx)) * WallNode->mu;  

               double U_w   = sqrt(tau_w/WallNode->S[i2d_Ro]);

               TmpNode->y_plus = U_w*TmpNode->l_min*TmpNode->S[i2d_Ro]/TmpNode->mu;
           }
        }
     }
  //__syncthreads(); 
}

__global__  void
cuda_DEEPS2D_Stage1(FlowNode2D<double,NUM_COMPONENTS>*     pLJ,
                    FlowNodeCore2D<double,NUM_COMPONENTS>* pLC,
                    unsigned long int index_limit,
                    int MAX_X, int MAX_Y,
                    unsigned long r_limit,
                    unsigned long l_limit,
                    double dxx, double dyy,
                    double dtdx, double dtdy,
                    double _dt,
                    int _FT, int Num_Eq,
                    SolverMode sm) {

    size_t index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

          FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pLJ[index];

          if(CurrentNode->ix <  r_limit &&
             CurrentNode->ix >= l_limit &&
             CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D) &&
             CurrentNode->CT != (ulong)(NT_FC_2D)) {
              
              FlowNodeCore2D< double,NUM_COMPONENTS >* NextNode=&pLC[index];

              double beta[NUM_COMPONENTS+6];
              double _beta[NUM_COMPONENTS+6];

              int  n1 = CurrentNode->idXl; 
              int  n2 = CurrentNode->idXr;
              int  n3 = CurrentNode->idYu;
              int  n4 = CurrentNode->idYd;

              double  n_n_1 = 1./max(n1+n2,1);
              double  m_m_1 = 1./max(n3+n4,1);

              FlowNode2D< double,NUM_COMPONENTS >* UpNode    = CurrentNode->UpNode;
              FlowNode2D< double,NUM_COMPONENTS >* DownNode  = CurrentNode->DownNode;
              FlowNode2D< double,NUM_COMPONENTS >* RightNode = CurrentNode->RightNode;
              FlowNode2D< double,NUM_COMPONENTS >* LeftNode  = CurrentNode->LeftNode;

              // Scan equation system ... k - number of equation
#pragma unroll
              for (int k=0;k<Num_Eq;k++ ) {

                  int      c_flag = 0;
                  int      dx_flag, dx2_flag;

                  int      dy_flag, dy2_flag;
                  double   dXX,dYY;

                  beta[k]  = CurrentNode->beta[k];
                  _beta[k] = 1. - beta[k];

                // Precomputed variables for current node ...
                    c_flag  = dx_flag = dy_flag = dx2_flag = dy2_flag = 0;
                    if ( k < 4 ) { // Make bit flags for future test for current equation
                        c_flag   = CT_Ro_CONST_2D     << k; 
                        dx_flag  = CT_dRodx_NULL_2D   << k;
                        dy_flag  = CT_dRody_NULL_2D   << k;
                        dx2_flag = CT_d2Rodx2_NULL_2D << k;
                        dy2_flag = CT_d2Rody2_NULL_2D << k;
                    } else if (k < (4+NUM_COMPONENTS)) {
                        c_flag   = CT_Y_CONST_2D;
                        dx_flag  = CT_dYdx_NULL_2D;
                        dy_flag  = CT_dYdy_NULL_2D;
                        dx2_flag = CT_d2Ydx2_NULL_2D;
                        dy2_flag = CT_d2Ydy2_NULL_2D;
                    } else if (sm == SM_NS &&
                               ((CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                                CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)))) { //
                      if( k == i2d_k) {
                          c_flag   = TCT_k_CONST_2D     << (k-4-NUM_COMPONENTS);
                          dx_flag  = TCT_dkdx_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dy_flag  = TCT_dkdy_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dx2_flag = TCT_d2kdx2_NULL_2D << (k-4-NUM_COMPONENTS);
                          dy2_flag = TCT_d2kdy2_NULL_2D << (k-4-NUM_COMPONENTS);
                      } else if (k == i2d_eps) {
                          c_flag   = TCT_eps_CONST_2D     << (k-4-NUM_COMPONENTS);
                          dx_flag  = TCT_depsdx_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dy_flag  = TCT_depsdy_NULL_2D   << (k-4-NUM_COMPONENTS);
                          dx2_flag = TCT_d2epsdx2_NULL_2D << (k-4-NUM_COMPONENTS);
                          dy2_flag = TCT_d2epsdy2_NULL_2D << (k-4-NUM_COMPONENTS);
                      }
                    }
                    // Check BC for current equation
                    if (k<(4+NUM_COMPONENTS)) {

                        if ( CurrentNode->isCond2D((CondType2D)c_flag) )
                            c_flag  = 0;
                        else
                            c_flag  = 1;

                        if ( CurrentNode->isCond2D((CondType2D)dx_flag) ) {
                            dx_flag = 0;
                        } else {
                            dx_flag = 1;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dy_flag) ) {
                            dy_flag = 0;
                        } else {
                            dy_flag = 1;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dx2_flag) ) {
                            dx2_flag = 1;
                        } else {
                            dx2_flag = 0;
                        }

                        if ( CurrentNode->isCond2D((CondType2D)dy2_flag) ) {
                            dy2_flag = 1;
                        } else {
                            dy2_flag = 0;
                        }
                    } else if(sm == SM_NS &&
                              (CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                               CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) ) {
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)c_flag) )
                            c_flag  = 0;
                        else
                            c_flag  = 1;

                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dx_flag) ) {
                            dx_flag = 0;
                        } else {
                            dx_flag = 1;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dy_flag) ) {
                            dy_flag = 0;
                        } else {
                            dy_flag = 1;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dx2_flag) ) {
                            dx2_flag = 1;
                        } else {
                            dx2_flag = 0;
                        }
                        if ( CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)dy2_flag) ) {
                            dy2_flag = 1;
                        } else {
                            dy2_flag = 0;
                        }
                    }

                    if ( c_flag ) {
                        if ( dx_flag ) {
                            dXX = CurrentNode->dSdx[k] = (RightNode->A[k]-LeftNode->A[k])*n_n_1; //  
                        } else {
                            CurrentNode->S[k] = (LeftNode->S[k]*n2+RightNode->S[k]*n1)*n_n_1;
                            dXX = CurrentNode->dSdx[k] = 0.;                                     //    
                        }
                        if ( dy_flag ) {
                            dYY = CurrentNode->dSdy[k] = (UpNode->B[k]-DownNode->B[k])*m_m_1;    // 
                        } else {
                            CurrentNode->S[k] =  (UpNode->S[k]*n3+DownNode->S[k]*n4)*m_m_1;
                            dYY = CurrentNode->dSdy[k] = 0;                                      // 
                        }

                        // Cauchy BC
                        
                        if ( dx2_flag ) {
                            dXX = (LeftNode->dSdx[k]+RightNode->dSdx[k])*0.5;
                        }
                        if ( dy2_flag ) {
                            dYY = (UpNode->dSdy[k]+DownNode->dSdy[k])*0.5;
                        }
                        
                        if ( _FT ) {
                            NextNode->S[k] = CurrentNode->S[k]*beta[k]+_beta[k]*(dxx*(LeftNode->S[k]+RightNode->S[k])+dyy*(UpNode->S[k]+DownNode->S[k]))*0.5
                                          - (dtdx*dXX+dtdy*(dYY+CurrentNode->F[k]/(CurrentNode->ix+0.5))) + (CurrentNode->Src[k])*_dt+CurrentNode->SrcAdd[k];
                        } else {
                            NextNode->S[k] = CurrentNode->S[k]*beta[k]+_beta[k]*(dxx*(LeftNode->S[k]+RightNode->S[k])+dyy*(UpNode->S[k]+DownNode->S[k]))*0.5
                                          - (dtdx*dXX+dtdy*dYY) + (CurrentNode->Src[k])*_dt+CurrentNode->SrcAdd[k];
                        }
                }
            }
       }
   }
}

__global__  void 
cuda_DEEPS2D_Stage2(FlowNode2D<double,NUM_COMPONENTS>*     pLJ,
                    FlowNodeCore2D<double,NUM_COMPONENTS>* pLC,
                    unsigned long int index_limit,
                    int MAX_X, int MAX_Y,
                    unsigned long r_limit,
                    unsigned long l_limit,
                    double beta_init, double  beta0, 
                    int b_FF, double CFL0,
                    ChemicalReactionsModelData2D* pCRMD,
                    int noTurbCond,
                    double SigW, double SigF, double dx_1, double dy_1, double delta_bl,
                    FlowType _FT, int Num_Eq,
#ifdef _RMS_
                    double*  RMS, 
                    int*     iRMS,
                    double   DD_max,
                    int*     i_c,
                    int*     j_c,
#endif // _RMS_
                    double* _Hu,
                    int _isSrcAdd,
                    unsigned int* dt_min_device, double int2float_scale,
                    TurbulenceExtendedModel TurbExtModel, 
                    SolverMode sm) {


    unsigned long int index = threadIdx.x + blockIdx.x * blockDim.x;

    if(index < index_limit) {

       FlowNode2D< double,NUM_COMPONENTS >* CurrentNode=&pLJ[index];

       if(CurrentNode->ix >= l_limit && 
          CurrentNode->ix <  r_limit &&  
          CurrentNode->CT != (ulong)(CT_SOLID_2D | CT_NODE_IS_SET_2D) &&
          CurrentNode->CT != (ulong)(NT_FC_2D) ) { 

              double  beta_min;

              beta_min = min(beta0,beta_init);

              FlowNodeCore2D< double,NUM_COMPONENTS >* NextNode=&pLC[index];

              int  n1 = CurrentNode->idXl; 
              int  n2 = CurrentNode->idXr;
              int  n3 = CurrentNode->idYu;
              int  n4 = CurrentNode->idYd;

              double  n_n_1 = 1./max(n1+n2,1);
              double  m_m_1 = 1./max(n3+n4,1);

              FlowNode2D< double,NUM_COMPONENTS >* UpNode    = CurrentNode->UpNode;
              FlowNode2D< double,NUM_COMPONENTS >* DownNode  = CurrentNode->DownNode;
              FlowNode2D< double,NUM_COMPONENTS >* RightNode = CurrentNode->RightNode;
              FlowNode2D< double,NUM_COMPONENTS >* LeftNode  = CurrentNode->LeftNode;

              double dx_1xn_n_1=dx_1*n_n_1;
              double dy_1xm_m_1=dy_1*m_m_1;

              // Scan equation system ... k - number of equation
#pragma unroll
              for (int k=0;k<Num_Eq;k++ ) {

                  int      c_flag = 0;

                  if ( k < 4 ) // Make bit flags for future test for current equation 
                      c_flag  = CT_Ro_CONST_2D   << k;
                  else if (k<(4+NUM_COMPONENTS))  // 7 ?
                      c_flag  = CT_Y_CONST_2D;
                  else if(sm == SM_NS &&
                          (CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                           CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D) )) 
                      c_flag  = TCT_k_CONST_2D << (k-4-NUM_COMPONENTS); 

                  if ( !CurrentNode->isCond2D((CondType2D)c_flag) && 
                        CurrentNode->S[k] != 0. ) {
                        double DD_local;
                        double Tmp;

                        if(k == i2d_RoU && k == i2d_RoV ) {
                            Tmp = sqrt(CurrentNode->S[i2d_RoU]*CurrentNode->S[i2d_RoU]+
                                       CurrentNode->S[i2d_RoV]*CurrentNode->S[i2d_RoV]+1.e-30); // Flux
                        } else {
                            Tmp = CurrentNode->S[k];
                        }

                        if(fabs(Tmp) > 1.e-15)
                           DD_local = fabs((NextNode->S[k]-CurrentNode->S[k])/Tmp);
                        else
                           DD_local = 0.0;

                        if( b_FF == BFF_L) {
                         //LINEAR locally adopted blending factor function  (LLABFF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+DD_local));
                         } else if( b_FF == BFF_LR) {
                         //LINEAR locally adopted blending factor function with relaxation (LLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+DD_local));
                         } else if( b_FF == BFF_S) {
                         //SQUARE locally adopted blending factor function (SLABF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+DD_local*DD_local));
                         } else if (b_FF == BFF_SR) {
                         //SQUARE locally adopted blending factor function with relaxation (SLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+DD_local*DD_local));
                         } else if( b_FF == BFF_SQR) {
                         //SQRT() locally adopted blending factor function (SQRLABF)
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+sqrt(DD_local)));
                         } else if( b_FF == BFF_SQRR) {
                         //SQRT() locally adopted blending factor function with relaxation (SQRLABFFR)
                           CurrentNode->beta[k] = min((beta_min+CurrentNode->beta[k])*0.5,(beta_min*beta_min)/(beta_min+sqrt(DD_local))); 
                         } else {
                           // Default->SQRLABF
                           CurrentNode->beta[k] = min(beta_min,(beta_min*beta_min)/(beta_min+sqrt(DD_local)));
                 }
#ifdef _RMS_
                         RMS[k+ii*Num_Eq] += DD_local;
                         iRMS[k+ii*Num_Eq]++;
                         DD_max[k+ii*Num_Eq] = max(DD_max[k+ii*Num_Eq],DD_local);

                         if ( DD_max[k+ii*Num_Eq] == DD_local ) {
                              i_c[ii] = i;
                              j_c[ii] = j;
                         }
#endif // RMS
                  }
                  if (k<(4+NUM_COMPONENTS)) {
                      if ( !CurrentNode->isCond2D((CondType2D)c_flag) )
                            CurrentNode->S[k]   = NextNode->S[k];
                  } else if (sm == SM_NS &&
                             (CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D) ||
                              CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) ){
                      if ( !CurrentNode->isTurbulenceCond2D((TurbulenceCondType2D)c_flag) )
                            CurrentNode->S[k]   =  NextNode->S[k];
                  }
              }

              CurrentNode->droYdx[NUM_COMPONENTS]=CurrentNode->droYdy[NUM_COMPONENTS]=0.;

#pragma unroll
              for (int k=4;k<NUM_EQ-2;k++ ) {
                  if ( !CurrentNode->isCond2D(CT_dYdx_NULL_2D) ) {
                      CurrentNode->droYdx[k-4]=(RightNode->S[k]-LeftNode->S[k])*dx_1*0.5;
                      CurrentNode->droYdx[NUM_COMPONENTS]+=(RightNode->S[k]-LeftNode->S[k])*dx_1*0.5;
                  }
                  if ( !CurrentNode->isCond2D(CT_dYdy_NULL_2D) ) {
                        CurrentNode->droYdy[k-4]=(UpNode->S[k]-DownNode->S[k])*dy_1*0.5;
                        CurrentNode->droYdy[NUM_COMPONENTS]+=(DownNode->S[k]-UpNode->S[k])*dy_1*0.5;
                  }
              }

              if (CurrentNode->isCond2D(CT_WALL_NO_SLIP_2D) || CurrentNode->isCond2D(CT_WALL_LAW_2D) )  {
                  CurrentNode->dUdx=(RightNode->U*n1-LeftNode->U*n2)*dx_1xn_n_1;
                  CurrentNode->dVdx=(RightNode->V*n1-LeftNode->V*n2)*dx_1xn_n_1;

                  CurrentNode->dUdy=(UpNode->U*n3-DownNode->U*n4)*dy_1xm_m_1;
                  CurrentNode->dVdy=(UpNode->V*n3-DownNode->V*n4)*dy_1xm_m_1;

                  if(sm == SM_NS && CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D)){
                    CurrentNode->dkdx   =(RightNode->S[i2d_k]*n1-LeftNode->S[i2d_k]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                    CurrentNode->depsdx =(RightNode->S[i2d_eps]*n1-LeftNode->S[i2d_eps]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;

                    CurrentNode->dkdy   =(UpNode->S[i2d_k]*n3-DownNode->S[i2d_k]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                    CurrentNode->depsdy =(UpNode->S[i2d_eps]*n3-DownNode->S[i2d_eps]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  } else if (sm == SM_NS && CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) {
                             CurrentNode->dkdx   =(RightNode->S[i2d_k]*n1-LeftNode->S[i2d_k]*n2)*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                             CurrentNode->dkdy   =(UpNode->S[i2d_k]*n3-DownNode->S[i2d_k]*n4)*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  }
              } else {
                  CurrentNode->dUdx   =(RightNode->U-LeftNode->U)*dx_1xn_n_1;
                  CurrentNode->dVdx   =(RightNode->V-LeftNode->V)*dx_1xn_n_1;

                  CurrentNode->dUdy   =(UpNode->U-DownNode->U)*dy_1xm_m_1;
                  CurrentNode->dVdy   =(UpNode->V-DownNode->V)*dy_1xm_m_1;
                  if(sm == SM_NS && CurrentNode->isTurbulenceCond2D(TCT_k_eps_Model_2D)){
                    CurrentNode->dkdx   =(RightNode->S[i2d_k]-LeftNode->S[i2d_k])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                    CurrentNode->depsdx =(RightNode->S[i2d_eps]-LeftNode->S[i2d_eps])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;

                    CurrentNode->dkdy   =(UpNode->S[i2d_k]-DownNode->S[i2d_k])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                    CurrentNode->depsdy =(UpNode->S[i2d_eps]-DownNode->S[i2d_eps])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  } else if (sm == SM_NS && CurrentNode->isTurbulenceCond2D(TCT_Spalart_Allmaras_Model_2D)) {
                             CurrentNode->dkdx   =(RightNode->S[i2d_k]-LeftNode->S[i2d_k])*dx_1/CurrentNode->S[i2d_Ro]*n_n_1;
                             CurrentNode->dkdy   =(UpNode->S[i2d_k]-DownNode->S[i2d_k])*dy_1/CurrentNode->S[i2d_Ro]*m_m_1;
                  }
              }
              
              CurrentNode->dTdx=(RightNode->Tg-LeftNode->Tg)*dx_1xn_n_1;
              CurrentNode->dTdy=(UpNode->Tg-DownNode->Tg)*dy_1xm_m_1;

              cuda_CalcChemicalReactions(CurrentNode,CRM_ZELDOVICH, (void*)(pCRMD));

              if(noTurbCond) {
                 CurrentNode->FillNode2D(0,1,SigW,SigF,TurbExtModel,delta_bl,1.0/dx_1,1.0/dy_1,_Hu,_isSrcAdd,sm,_FT);
              } else {
                 CurrentNode->FillNode2D(1,0,SigW,SigF,TurbExtModel,delta_bl,1.0/dx_1,1.0/dy_1,_Hu,_isSrcAdd,sm,_FT);
              }

              if( CurrentNode->Tg < 0. ) {
                  *dt_min_device = 0;  // Computational instability
              }  else {
                  double AAA          = sqrt(CurrentNode->k*CurrentNode->R*CurrentNode->Tg); 
                  double dt_min_local = CFL0*min(1.0/(dx_1*(AAA+fabs(CurrentNode->U))),1.0/(dy_1*(AAA+fabs(CurrentNode->V))));
                  atomicMin(dt_min_device,(unsigned int)(dt_min_local*int2float_scale));
              }
         }
      }
   }
#endif // _CUDA_

